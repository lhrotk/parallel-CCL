
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <map>
#include <queue>
#include <list>
#include <algorithm>
#include <utility>
#include <cmath>
#include <functional>
#include <cstring>
#include <cmath>
#include <limits>



#define NOMINMAX

#ifdef _MSC_VER
#include <ctime>
inline double get_time()
{
        return static_cast<double>(std::clock()) / CLOCKS_PER_SEC;
}
#else
#include <sys/time.h>
inline double get_time()
{
        timeval tv;
        gettimeofday(&tv, 0);
        return tv.tv_sec + 1e-6 * tv.tv_usec;
}
#endif

using namespace std;

//const int BLOCK = 128;
const int BLOCK = 1024;

__global__ void init_CCL(int L[], int N)
{
	for(int id = blockIdx.x * blockDim.x + threadIdx.x; id<N; id+=blockDim.x*gridDim.x){
		L[id] = id;
	}
}

__global__ void scanning(int D[], int L[], int N, int W, int th)
{
	for(int id = blockIdx.x * blockDim.x + threadIdx.x; id<N; id+=blockDim.x*gridDim.x){
		if(D[id]==0)
			L[id]=0;
		else{
			if(id-W>=0 && D[id-W]!=0)
				L[id] = min(L[id], L[id-W]);
			if(id%W!=N && D[id-1]!=0)
				L[id] = min(L[id], L[id-1]);
		}
	}
}

__global__ void analysis(int D[], int L[], int W, int N)
{
	for(int id = blockIdx.x * blockDim.x + threadIdx.x; id<N; id+=blockDim.x*gridDim.x){
		int ref;
		if(D[id]!=0&&id-W>=0 && D[id-W]!=0&&id%W!=0 && D[id-1]!=0){
			int label1 = L[id-1];
			int label2 = L[id-W];
			do { label1 = L[ref = label1]; } while (ref ^ label1);
			do { label2 = L[ref = label2]; } while (ref ^ label2);
			if(label1>label2){
				atomicAdd(L+label1, label2-label1);
			}else if(label1<label2){
				atomicAdd(L+label2, label1-label2);
			}else{
				L[id] = label1;
			}
		}else{
			int label1 = L[id];
			do { label1 = L[ref = label1]; } while (ref ^ label1);
			L[id] = label1;
		}
	}
}

__global__ void labeling(int D[], int L[], int N)
{
	for (int id = blockIdx.x * blockDim.x + threadIdx.x; id<N; id+=blockDim.x*gridDim.x) {
		int label = L[id];
		int ref;
		do { label = L[ref = label]; } while (ref != label);
		L[id] = label;
	}
}

class CCL {
private:
	int* Dd;
	int* Ld;
	int* Rd;

public:
	vector<int> cuda_ccl(vector<int>& image, int W, int degree_of_connectivity, int threshold);
};

vector<int> CCL::cuda_ccl(vector<int>& image, int W, int degree_of_connectivity, int threshold)
{
	int* D = static_cast<int*>(&image[0]);
	int N = image.size();
	int* Label = new int[N];
	hipMalloc((void**)&Ld, sizeof(int) * N);
	//cudaMalloc((void**)&Rd, sizeof(int) * N);
	hipMalloc((void**)&Dd, sizeof(int) * N);

	
	dim3 grid(6, 1, 1);
	dim3 threads(1024, 1, 1);
	double start = get_time();
	
	hipMemcpy(Dd, D, sizeof(int) * N, hipMemcpyHostToDevice);

	
	init_CCL<<<grid, threads>>>(Ld, N);
	scanning<<<grid, threads>>>(Dd, Ld, N, W, threshold);
	analysis<<<grid, threads>>>(Dd, Ld, W, N);
	labeling<<<grid, threads>>>(Dd, Ld, N);
	hipDeviceSynchronize();
	hipMemcpy(Label, Ld, sizeof(int) * N, hipMemcpyDeviceToHost);
	
	double end = get_time();
	cerr << "Time: " << end - start << endl;


	hipFree(Dd);
	hipFree(Ld);
	hipFree(Rd);
	vector<int> result(Label, Label + N);

	delete [] Label;
	return result;
}

void read_data(const string filename, vector<int>& image, int& W, int& degree_of_connectivity, int& threshold)
{
	fstream fs(filename.c_str(), ios_base::in);
	string line;
	stringstream ss;
	int data;

	getline(fs, line);
	ss.str(line);
	ss >> W >> degree_of_connectivity >> threshold;
	getline(fs, line);
	ss.str("");  ss.clear();
	for (ss.str(line); ss >> data; image.push_back(data));
}

int main(int argc, char* argv[])
{
	ios_base::sync_with_stdio(false);

	if (argc < 2) {
		cerr << "Usage: " << argv[0] << " input_file" << endl;
		exit(1);
	}

	vector<int> image;
	int W, degree_of_connectivity, threshold;
	read_data(argv[1], image, W, degree_of_connectivity, threshold);

	CCL ccl;

	//double start = get_time();
	vector<int> result(ccl.cuda_ccl(image, W, degree_of_connectivity, threshold));
	//double end = get_time();
	//cerr << "Time: " << end - start << endl;

	cout << result.size() << endl; /// number of pixels
	cout << W << endl; /// width
	bool judge = true;
	for (int i = 0; i < image.size()/W; i++) {
		for (int j = 0; j < W; j++) cout << result[i*W+j] << " ";
		cout << endl;
	}
	for (int i = 0; i < image.size()/W-1; i++) {
		for (int j = 0; j < W-1; j++){
			if(image[i*W+j]==255&&image[i*W+W+j]==255&&result[i*W+W+j]!=result[i*W+j]){
				judge = false;
				cerr << i <<" "<< j << " " << i+1 <<" "<< j << endl;
			}
			if(image[i*W+j]==255&&image[i*W+1+j]==255&&result[i*W+1+j]!=result[i*W+j]){
				judge = false;
				cerr << i <<" "<< j << " " << i <<" "<< j+1 << endl;
			}
		}
	}
	if(judge)
		cerr << "result is correct" << endl;
	return 0;
}
